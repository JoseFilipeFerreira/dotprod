/**************************************************************
 *
 * --== Simple CUDA kernel ==--
 * author: ampereira
 *
 *
 * Fill the rest of the code
 *
 * Insert the functions for time measurement in the correct
 * sections (i.e. do not account for filling the vectors with random data)
 *
 *
 * The dot array size must be set to the SIZE #define, i.e., float dot[SIZE];
 **************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <sys/time.h>

#define TIME_RESOLUTION 1000000	// time measuring resolution (us)

#define NUM_BLOCKS 128
#define NUM_THREADS_PER_BLOCK 128
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK
#define NEIGHBOURS 4

using namespace std;

long long unsigned cpu_time;
hipEvent_t start, stop;
timeval t;

void startTime (void) {
    gettimeofday(&t, NULL);
    cpu_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;
}

void stopTime (void) {
    gettimeofday(&t, NULL);
    long long unsigned final_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;

    final_time -= cpu_time;

    cout << final_time << " us have elapsed for the CPU execution" << endl;
}

// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime (void) {
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
}

void stopKernelTime (void) {
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << milliseconds << " ms have elapsed for the CUDA execution" << endl;
}

void checkCUDAError (const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
        exit(-1);
    }
}

// Fill the input parameters and kernel qualifier
__global__
void dotKernel (float *dev_m1, float *dev_m2, float *dev_output) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    //block = blockId * blockDim - 0 -> max threadId
    int i = SIZE / id;
    int k = SIZE % id;
    for (size_t j = 0; k < NUM_THREADS_PER_BLOCK; k++) {
        dev_output[blockIdx.x*blockDim.x+j] += dev_m1[id] * dev_m2[k*blockDim.x+j];
    }
}

// Fill with the code required for the GPU dot (mem allocation, transfers, kernel launch of dotKernel)
float* dotGPU (float *m1, float *m2) {
    float *dev_m1, *dev_m2, *dev_output;
    float *array_output = new float [SIZE];

    // allocate the memory on the device
    hipMalloc((void**) &dev_m1, sizeof(float) * SIZE);
    hipMalloc((void**) &dev_m2, sizeof(float) * SIZE);
    hipMalloc((void**) &dev_output, sizeof(float) * SIZE);

    startKernelTime();
    // copy inputs to the device
    hipMemcpy(dev_m1, m1, sizeof(float) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_m2, m2, sizeof(float) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_output, array_output, sizeof(float) * SIZE, hipMemcpyHostToDevice);

    // launch the kernel
    dotKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>> (dev_m1, dev_m2, dev_output);

    // copy the output to the host
    hipMemcpy(array_output, dev_output, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
    stopKernelTime();
    for(size_t i = 0; i < SIZE; i++)
        cout << array_output[i] << '\n';

    // free the device memory
    hipFree(dev_m1);
    hipFree(dev_m2);
    hipFree(dev_output);

    return array_output;
}

int main (int argc, char** argv) {
    float array1 [SIZE];
    float array2 [SIZE];
    // initialize array with random values
    for (unsigned i = 0; i < SIZE; i++) {
        array1[i] = ((float) rand()) / ((float) RAND_MAX);
        array2[i] = ((float) rand()) / ((float) RAND_MAX);
    }
    dotGPU(array1, array2);
    return 0;
}
