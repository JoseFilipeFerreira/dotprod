
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <sys/time.h>

#define TIME_RESOLUTION 1000000	// time measuring resolution (us)

#define NUM_BLOCKS 128
#define NUM_THREADS_PER_BLOCK 128
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK
#define NEIGHBOURS 4

using namespace std;

long long unsigned cpu_time;
hipEvent_t start, stop;
timeval t;

void startTime (void) {
    gettimeofday(&t, NULL);
    cpu_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;
}

void stopTime (void) {
    gettimeofday(&t, NULL);
    long long unsigned final_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;

    final_time -= cpu_time;

    cout << final_time << " us have elapsed for the CPU execution" << endl;
}

// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime (void) {
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
}

void stopKernelTime (void) {
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << milliseconds << " ms have elapsed for the CUDA execution" << endl;
}

void checkCUDAError (const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
        exit(-1);
    }
}

// Fill the input parameters and kernel qualifier
__global__
void dotKernel (float *dev_m1, float *dev_m2, float *dev_output) {
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ float shareA[NUM_BLOCKS/16][NUM_BLOCKS/16];
    __shared__ float shareB[NUM_BLOCKS/16][NUM_BLOCKS/16];
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    float tmp_sum = 0.0f;
    for(int i = 0; i < blockDim.x; ++i){
        shareA[ty][tx] = dev_m1[row*blockDim.x + (i*2 + tx)];
        shareB[ty][tx] = dev_m2[(i*2 + ty)*blockDim.x + col];
        __syncthreads();
        for(int k = 0; k < blockDim.x; ++k){
            tmp_sum += shareA[ty][k] * shareB[k][tx];
            __syncthreads();
        }
        dev_output[ROW * NUM_BLOCKS + COL] = tmp_sum;
    }
    /*
    if (ROW < NUM_BLOCKS && COL < NUM_BLOCKS) {
        float tmp_sum = 0.0f;
        for (int i = 0; i < NUM_BLOCKS; i++) {
            tmp_sum += dev_m1[ROW * NUM_BLOCKS + i] * dev_m2[i * NUM_BLOCKS + COL];
        }
        dev_output[ROW * NUM_BLOCKS + COL] = tmp_sum;
    }
    */
}

// Fill with the code required for the GPU dot (mem allocation, transfers, kernel launch of dotKernel)
float* dotGPU (float *m1, float *m2) {
    float *dev_m1, *dev_m2, *dev_output;
    float *array_output = new float [SIZE];

    // allocate the memory on the device
    hipMalloc((void**) &dev_m1, sizeof(float) * SIZE);
    hipMalloc((void**) &dev_m2, sizeof(float) * SIZE);
    hipMalloc((void**) &dev_output, sizeof(float) * SIZE);

    startKernelTime();
    // copy inputs to the device
    hipMemcpy(dev_m1, m1, sizeof(float) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_m2, m2, sizeof(float) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_output, array_output, sizeof(float) * SIZE, hipMemcpyHostToDevice);

    dim3 blocksPerGrid(16, 16, 1);
    dim3 threadsPerBlock(NUM_THREADS_PER_BLOCK/16, NUM_THREADS_PER_BLOCK/16, 1);
    // launch the kernel
    dotKernel <<< blocksPerGrid, threadsPerBlock >>> (dev_m1, dev_m2, dev_output);

    // copy the output to the host
    hipMemcpy(array_output, dev_output, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
    stopKernelTime();
    for(size_t i = 0; i < 128; i++) {
        for(size_t j = 0; j < 128; j++)
            cout << array_output[i*128 + j] << ' ';
        cout << '\n';

    }
    // free the device memory
    hipFree(dev_m1);
    hipFree(dev_m2);
    hipFree(dev_output);

    return array_output;
}

int main (int argc, char** argv) {
    float array1 [SIZE];
    float array2 [SIZE];
    // initialize array with random values
    for (unsigned i = 0; i < SIZE; i++) {
        array1[i] = ((float) rand()) / ((float) RAND_MAX);
        array2[i] = 1;
    }
    dotGPU(array1, array2);
    return 0;
}
