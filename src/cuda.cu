
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <vector>

#define BS 32
#define NUM_BLOCKS 1500
#define NUM_THREADS_PER_BLOCK 1500
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK

using namespace std;

hipEvent_t start, stop;

// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime (void) {
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
}

void stopKernelTime (void) {
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << milliseconds << " ms have elapsed for the CUDA execution" << endl;
}

void checkCUDAError (const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
        exit(-1);
    }
}

// Fill the input parameters and kernel qualifier
__global__
void dotKernel (float *dev_m1, float *dev_m2, float *dev_output) {
    __shared__ float shareA[BS][BS];
    __shared__ float shareB[BS][BS];
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int row = by * BS + ty;
    int col = bx * BS + tx;
    float tmp_sum = 0.0f;
    for(int i = 0; i < NUM_BLOCKS/BS; ++i){
        shareA[ty][tx] = dev_m1[row* NUM_BLOCKS + (i * BS + tx)];
        shareB[ty][tx] = dev_m2[(i * BS + ty) * NUM_BLOCKS + col];
        __syncthreads();
        for(int k = 0; k < BS; ++k){
            tmp_sum += shareA[ty][k] * shareB[k][tx];
            __syncthreads();
        }
        dev_output[row * NUM_BLOCKS + col] = tmp_sum;
    }
}

// Fill with the code required for the GPU dot (mem allocation, transfers, kernel launch of dotKernel)
float* dotGPU (float *m1, float *m2) {
    float *dev_m1, *dev_m2, *dev_output;
    float *array_output = new float [SIZE];

    // allocate the memory on the device
    hipMalloc((void**) &dev_m1, sizeof(float) * SIZE);
    hipMalloc((void**) &dev_m2, sizeof(float) * SIZE);
    hipMalloc((void**) &dev_output, sizeof(float) * SIZE);

    startKernelTime();
    // copy inputs to the device
    hipMemcpy(dev_m1, m1, sizeof(float) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_m2, m2, sizeof(float) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_output, array_output, sizeof(float) * SIZE, hipMemcpyHostToDevice);

    dim3 blocksPerGrid(BS, BS, 1);
    dim3 threadsPerBlock(NUM_THREADS_PER_BLOCK/BS, NUM_THREADS_PER_BLOCK/BS, 1);
    // launch the kernel
    dotKernel <<< blocksPerGrid, threadsPerBlock >>> (dev_m1, dev_m2, dev_output);

    // copy the output to the host
    hipMemcpy(array_output, dev_output, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
    stopKernelTime();
    
    for(size_t i = 0; i < 512; i++) {
        cout << array_output[i] << '\n';
    }
    // free the device memory
    hipFree(dev_m1);
    hipFree(dev_m2);
    hipFree(dev_output);

    return array_output;
}

int main (int argc, char** argv) {
    float array1 [SIZE];
    float array2 [SIZE];
    // initialize array with random values
    for (unsigned i = 0; i < SIZE; i++) {
        array1[i] = ((float) rand()) / ((float) RAND_MAX) * 10;
        array2[i] = 1;
    }
    for(size_t i = 0; i < 4; i++) {
        cout << array1[i * NUM_BLOCKS] << '\n';
    }
    dotGPU(array1, array2);
    return 0;
}
